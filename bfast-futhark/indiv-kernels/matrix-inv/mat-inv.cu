
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


__global__ void
writeShapeKernel(float** Ashp, float* Aflat, int n, int M) {
    const unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if(gid < M) {
        Ashp[gid] = Aflat + gid*n*n;
    }
}

static int64_t get_wall_time(void) {
  struct timeval time;
  gettimeofday(&time,NULL);
  return time.tv_sec * 1000000 + time.tv_usec;
}

void invert0(float** src, float** dst, int n, int batchSize)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int *P, *INFO;

    cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
    cudacall(hipMalloc(&INFO,  batchSize * sizeof(int)));

    int lda = n;

    float **A = (float **)malloc(batchSize*sizeof(float *));
    float **A_d, *A_dflat;
    cudacall(hipMalloc(&A_d,batchSize*sizeof(float *)));
    cudacall(hipMalloc(&A_dflat, n*n*batchSize*sizeof(float)));
    A[0] = A_dflat;
    for (int i = 1; i < batchSize; i++)
      A[i] = A[i-1]+(n*n);
    cudacall(hipMemcpy(A_d,A,batchSize*sizeof(float *),hipMemcpyHostToDevice));
    for (int i = 0; i < batchSize; i++)
      cudacall(hipMemcpy(A_dflat+(i*n*n), src[i], n*n*sizeof(float), hipMemcpyHostToDevice));

    cublascall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh[batchSize];
    cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

    for (int i = 0; i < batchSize; i++)
      if(INFOh[i]  != 0)
      {
        fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
        hipDeviceReset();
        exit(EXIT_FAILURE);
      }

    float **C = (float **)malloc(batchSize*sizeof(float *));
    float **C_d, *C_dflat;
    cudacall(hipMalloc(&C_d,batchSize*sizeof(float *)));
    cudacall(hipMalloc(&C_dflat, n*n*batchSize*sizeof(float)));
    C[0] = C_dflat;
    for (int i = 1; i < batchSize; i++)
      C[i] = C[i-1] + (n*n);
    cudacall(hipMemcpy(C_d,C,batchSize*sizeof(float *),hipMemcpyHostToDevice));
    cublascall(hipblasSgetriBatched(handle,n,(const float **)A_d,lda,P,C_d,lda,INFO,batchSize));

    cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

    for (int i = 0; i < batchSize; i++)
      if(INFOh[i] != 0)
      {
        fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
        hipDeviceReset();
        exit(EXIT_FAILURE);
      }
    for (int i = 0; i < batchSize; i++)
      cudacall(hipMemcpy(dst[i], C_dflat + (i*n*n), n*n*sizeof(float), hipMemcpyDeviceToHost));
    hipFree(A_d); hipFree(A_dflat); free(A);
    hipFree(C_d); hipFree(C_dflat); free(C);
    hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}


void invert(float** src, float** dst, int n, int batchSize)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int *P, *INFO;

    int lda = n;

    float **A = (float **)malloc(batchSize*sizeof(float *));
    float **A_d, *A_dflat;
    cudacall(hipMalloc(&A_d,batchSize*sizeof(float *)));
    cudacall(hipMalloc(&A_dflat, n*n*batchSize*sizeof(float)));
    A[0] = A_dflat;
    for (int i = 1; i < batchSize; i++)
      A[i] = A[i-1]+(n*n);
    cudacall(hipMemcpy(A_d,A,batchSize*sizeof(float *),hipMemcpyHostToDevice));
    for (int i = 0; i < batchSize; i++)
      cudacall(hipMemcpy(A_dflat+(i*n*n), src[i], n*n*sizeof(float), hipMemcpyHostToDevice));


    // for second call
    float **C = (float **)malloc(batchSize*sizeof(float *));
    float **C_d, *C_dflat;
    cudacall(hipMalloc(&C_d,batchSize*sizeof(float *)));
    cudacall(hipMalloc(&C_dflat, n*n*batchSize*sizeof(float)));
    C[0] = C_dflat;
    for (int i = 1; i < batchSize; i++)
      C[i] = C[i-1] + (n*n);
    cudacall(hipMemcpy(C_d,C,batchSize*sizeof(float *),hipMemcpyHostToDevice));

    int INFOh[batchSize];


    {
        hipDeviceSynchronize();
        int64_t elapsed, aft, bef = get_wall_time();
        cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
        cudacall(hipMalloc(&INFO,  batchSize * sizeof(int)));

        const unsigned int block_size = 256;
        const unsigned int num_blocks = (batchSize + block_size - 1) / block_size;
        
        writeShapeKernel<<<num_blocks,block_size>>>(A_d, A_dflat, n, batchSize);
        cublascall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));
#if 0
        cudacall(cudaMemcpy(INFOh,INFO,batchSize*sizeof(int),cudaMemcpyDeviceToHost));

        for (int i = 0; i < batchSize; i++) {
            if(INFOh[i]  != 0)
            {
                fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
                cudaDeviceReset();
                exit(EXIT_FAILURE);
            }
        }
#endif        
        writeShapeKernel<<<num_blocks,block_size>>>(C_d, C_dflat, n, batchSize);
        cublascall(hipblasSgetriBatched(handle,n,(const float **)A_d,lda,P,C_d,lda,INFO,batchSize));
        hipDeviceSynchronize();

        aft = get_wall_time();
        elapsed = aft - bef;
        printf("%ldμs\n", elapsed);
    }


    cudacall(hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost));

    for (int i = 0; i < batchSize; i++)
      if(INFOh[i] != 0)
      {
        fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
        hipDeviceReset();
        exit(EXIT_FAILURE);
      }
    for (int i = 0; i < batchSize; i++)
      cudacall(hipMemcpy(dst[i], C_dflat + (i*n*n), n*n*sizeof(float), hipMemcpyDeviceToHost));
    hipFree(A_d); hipFree(A_dflat); free(A);
    hipFree(C_d); hipFree(C_dflat); free(C);
    hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}


float** mkRandData(int K, int M) {
    float **inputs = (float **)malloc(M*sizeof(float *));
    for(int i=0; i<M; i++) {
        float* mat = (float*)malloc(K*K*sizeof(float));
        for(int k=0; k<K*K; k++) {
            mat[k] = (rand() / (float)RAND_MAX) * 1000.0;
        }
        inputs[i] = mat;
    }
    return inputs;
}


void test_invert(const int mybatch, const int n)
{
#if 0
    const int n = 8; //3; //8;
    const int mybatch = 111556; //16384*4; //4; //16384;

    //Random matrix with full pivots
    float full_pivot[n*n] = { 0.5, 3, 4,
                                1, 3, 10,
                                4 , 9, 16 };

    //Almost same as above matrix with first pivot zero
    float zero_pivot[n*n] = { 0, 3, 4,
                              1, 3, 10,
                              4 , 9, 16 };

    float another_zero_pivot[n*n] = { 0, 3, 4,
                                      1, 5, 6,
                                      9, 8, 2 };

    float another_full_pivot[n * n] = { 22, 3, 4,
                                        1, 5, 6,
                                        9, 8, 2 };

    float **inputs = (float **)malloc(mybatch*sizeof(float *));
    inputs[0]  = zero_pivot;
    inputs[1]  = full_pivot;
    inputs[2]  = another_zero_pivot;
    inputs[3]  = another_full_pivot;
#else
    float** inputs = mkRandData(n, mybatch);
#endif


    float *result_flat = (float *)malloc(mybatch*n*n*sizeof(float));
    float **results = (float **)malloc(mybatch*sizeof(float *));
    for (int i = 0; i < mybatch; i++)
      results[i] = result_flat + (i*n*n);

#if 0
    for (int qq = 0; qq < mybatch; qq++){
      fprintf(stdout, "Input %d:\n\n", qq);
      for(int i=0; i<n; i++)
      {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",inputs[qq][i*n+j]);
        fprintf(stdout,"\n");
      }
    }
    fprintf(stdout,"\n\n");
#endif 

    invert(inputs, results, n, mybatch);

#if 0
    for (int qq = 0; qq < mybatch; qq++){
      fprintf(stdout, "Inverse %d:\n\n", qq);
      for(int i=0; i<n; i++)
      {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",results[qq][i*n+j]);
        fprintf(stdout,"\n");
      }
    }
#endif
}

int main(int argc, char** argv)
{
    if(argc != 3) {
        printf("Mat-Inv expects 2 arguments:\n");
        printf("(1) the size of the batch\n");
        printf("(2) the dimension K of the KxK matrix\n");
        exit(0);
    }
    int32_t M = atoi(argv[1]);
    int32_t K = atoi(argv[2]);
    test_invert(M, K);

    return 0;
}
